
#include <hip/hip_runtime.h>
#include <stdio.h>

// no input var needed by setting (void)
__global__ void whoami(void) {
    int block_id = 
        blockIdx.x + 
        blockIdx.y * gridDim.x +
        blockIdx.z * gridDim.x * gridDim.y;

    int block_offset = 
        block_id * // times our apartment number
        blockDim.x * blockDim.y * blockDim.z; // total threads per block (people per apartment)

    int thread_offset =
        threadIdx.x +  
        threadIdx.y * blockDim.x +
        threadIdx.z * blockDim.x * blockDim.y;

    int id = block_offset + thread_offset; // global person id in the entire apartment complex

    printf("%04d | Block(%d %d %d) = %3d | Thread(%d %d %d) = %3d\n",
        id,
        blockIdx.x, blockIdx.y, blockIdx.z, block_id,
        threadIdx.x, threadIdx.y, threadIdx.z, thread_offset);

}


int main(int argc, char **argv) {
    const int b_x = 2, b_y = 3, b_z = 4;
    const int t_x = 4, t_y = 4, t_z = 4; // the max warp size is 32, so 
    // we will get 2 warp of 32 threads per block

    int blocks_per_grid = b_x * b_y * b_z;
    int threads_per_block = t_x * t_y * t_z;

    printf("%d blocks/grid\n", blocks_per_grid);
    printf("%d threads/block\n", threads_per_block);
    printf("%d total threads\n", blocks_per_grid * threads_per_block);

    dim3 blocksPerGrid(b_x, b_y, b_z); // 3d cube of shape 2*3*4 = 24, a grid with (2,3,4) blocks
    dim3 threadsPerBlock(t_x, t_y, t_z); // 3d cube of shape 4*4*4 = 64, a block with (4,4,4) threads

    whoami<<<blocksPerGrid, threadsPerBlock>>>();
    hipDeviceSynchronize();
}